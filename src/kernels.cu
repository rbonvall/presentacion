
#include <hip/hip_runtime.h>
__global__ void set_one(float *array, int i) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id == 0) {
        array[i] = 1.0f;
    }
}

__device__ float gpu_logistic(float x) {
    return 1 / (1 + expf(-x));
}

__global__ void activation_function(float x[], int n) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n){
        x[id] = gpu_logistic(x[id]);
    }
}
