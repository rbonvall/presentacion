void matrix_vector_product(int nr_rows, int nr_cols,
                           float *b,
                           float *a,
                           float *x) {
    hipblasSgemv('t', nr_cols, nr_rows,
      1.0f, a, nr_cols,
            x, 1,
      0.0f, b, 1);
}
